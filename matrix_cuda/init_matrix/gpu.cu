#include "hip/hip_runtime.h"
#include "mmult.h"

__global__ void mmult_kernel(int m, int n, int k, const float * a, const float * b, float * c)
{
	int globx = blockIdx.x * blockDim.x + threadIdx.x;
	int globy = blockIdx.y * blockDim.y + threadIdx.y;

	__shared__ int l;

	for (l = 0; l < n; l++)
		c[globx * k + globy] += a[globx * n + l] * b[l * k + globy];
}


void mmult_gpu(int m, int n, int k, const float * a, const float * b, float * c)
{
	dim3 dim_Grid(m/BLOCK_SIZE, k/BLOCK_SIZE);
	dim3 dim_Block(BLOCK_SIZE,BLOCK_SIZE);
	mmult_kernel<<<dim_Grid, dim_Block>>>(m, n, k, a, b, c);
}
